#include "hip/hip_runtime.h"
#include "cusp/hyb_matrix.h"
#include "cusp/csr_matrix.h"
#include "cusp/gallery/poisson.h"
#include "cusp/krylov/cg.h"
#include <cusp/precond/diagonal.h>

void genLaplace(cusp::csr_matrix<int, float, cusp::host_memory>* A, int N, int nz, cusp::array1d<float, cusp::host_memory>* rhs, float wFit, float wReg, float* target)
{
	int n = (int)sqrt((double)N);
	std::cout << n << std::endl;
	int idx = 0;
	for (int i = 0; i<N; i++)
	{
		int ix = i%n;
		int iy = i / n;

		A->row_offsets[i] = idx;

		int count = 0;
		if (iy > 0)		count++;
		if (ix > 0)		count++;
		if (ix < n - 1) count++;
		if (iy < n - 1) count++;

		// up
		if (iy > 0)
		{
			A->values[idx] = -1.0*wReg;
			A->column_indices[idx] = i - n;
			idx++;
		}

		// left
		if (ix > 0)
		{
			A->values[idx] = -1.0*wReg;
			A->column_indices[idx] = i - 1;
			idx++;
		}

		// center
		A->values[idx] = count*wReg + wFit;
		A->column_indices[idx] = i;
		idx++;

		(*rhs)[i] = wFit*target[iy*n + ix];

		//right
		if (ix  < n - 1)
		{
			A->values[idx] = -1.0*wReg;
			A->column_indices[idx] = i + 1;
			idx++;
		}

		//down
		if (iy  < n - 1)
		{
			A->values[idx] = -1.0*wReg;
			A->column_indices[idx] = i + n;
			idx++;
		}
	}

	A->row_offsets[N] = idx;
}

int solve(unsigned int N, unsigned int nz, float wFit, float wReg, float* target)
{
	cusp::csr_matrix<int, float, cusp::host_memory> A_CPU(N, N, nz);
	cusp::array1d<float, cusp::host_memory>			b_CPU(N, 0);

	genLaplace(&A_CPU, N, nz, &b_CPU, wFit, wReg, target);

	A_CPU.row_offsets[0];
	A_CPU.column_indices[0]; A_CPU.values[0] = 10;

	cusp::csr_matrix<int, float, cusp::device_memory>	A_GPU(A_CPU);
	cusp::precond::diagonal<float, cusp::device_memory> M_GPU(A_GPU);
	cusp::array1d<float, cusp::device_memory>			b_GPU(b_CPU);
	cusp::array1d<float, cusp::device_memory>			x_GPU(N, 0);

	cusp::verbose_monitor<float> monitor(b_GPU,100, 0, 0);
	cusp::krylov::cg(A_GPU, x_GPU, b_GPU, monitor, M_GPU);

	return 0;
}
