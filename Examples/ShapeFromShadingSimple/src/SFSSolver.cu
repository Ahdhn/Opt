#include "hip/hip_runtime.h"
#include <iostream>

#include "SFSSolverParameters.h"
#include "SFSSolverState.h"
#include "SFSSolverState.h"
#include "SFSSolverUtil.h"
#include "SFSSolverEquations.h"

#include <assert.h>
#include <stdio.h>
#include <stdint.h>

#include "ConvergenceAnalysis.h"
#include "CUDATimer.h"

#ifdef _WIN32
#include <conio.h>
#endif

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

#define WARP_SIZE 32u
#define WARP_MASK (WARP_SIZE-1u)

#define DEBUG_PRINT_INFO 0


/*
static void checkForNan(std::string name, float* cudaPtr, int W, int H) {
int numBytes = W*H*sizeof(float);
float* ptr = (float*)malloc(numBytes);
printf("%s:\n", name.c_str());
cutilSafeCall(hipMemcpy(ptr, cudaPtr, numBytes, hipMemcpyDeviceToHost));

for (int i = 0; i < W*H; ++i) {
if (isnan(ptr[i])) {
printf("Is nan at (%d, %d)\n", i % W, i / W);
}
}
free(ptr);

}
*/

static void checkEverythingForNan(SolverInput& input, SolverState& state) {
    /*
    checkForNan("d_x", state.d_x, input.width, input.height);
    checkForNan("d_preconditioner", state.d_preconditioner, input.width, input.height);

    checkForNan(state.d_delta,          input.width, input.height);
    checkForNan(state.d_r,              input.width, input.height);
    checkForNan(state.d_z,              input.width, input.height);
    checkForNan(state.d_p,              input.width, input.height);
    //checkForNan(state.d_Ap_X,           input.width, input.height);
    //checkForNan(state.d_scanAlpha,      input.width, input.height);
    //checkForNan(state.d_scanBeta,       input.width, input.height);
    checkForNan(state.d_rDotzOld,       input.width, input.height);
    checkForNan(state.d_preconditioner, input.width, input.height);
    //m_solverState.d_sumResidual,    sizeof(float)));
    */


}

/////////////////////////////////////////////////////////////////////////
// Eval Residual
////////////////////////  /////////////////////////////////////////////////

__global__ void ResetResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x == 0) state.d_sumResidual[0] = 0.0f;
}

__global__ void EvalResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    const unsigned int N = input.N;
    

    float residual = 0.0f;
	if (x < N)
	{
		residual = evalFDevice(x, input, state, parameters);
	}
    // Must do shuffle in entire warp
    float r = warpReduce(residual);
    if ((threadIdx.x & WARP_MASK) == 0) {
        atomicAdd(state.d_sumResidual, r);
    }
}

float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	float residual = 0.0f;

	const unsigned int N = input.N; // Number of block variables
	ResetResidualDevice << < 1, 1, 1 >> >(input, state, parameters);
	cutilSafeCall(hipDeviceSynchronize());
	timer.startEvent("EvalResidual");
	EvalResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
	timer.endEvent();
	cutilSafeCall(hipDeviceSynchronize());

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

    cutilSafeCall(hipMemcpy(&residual, &state.d_sumResidual[0], sizeof(float), hipMemcpyDeviceToHost));

	return residual;
}

// For the naming scheme of the variables see:
// http://en.wikipedia.org/wiki/Conjugate_gradient_method
// This code is an implementation of their PCG pseudo code

__global__ void PCGInit_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
    const unsigned int N = input.N;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    float d = 0.0f;
    if (x < N)
    {
        float pre = 1.0f;
        float residuum = evalMinusJTFDevice(x, input, state, parameters, pre); // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 
        residuum = 2.0f * residuum;//TODO: Check if results are still okay once we fix this
        
        state.d_r[x] = residuum;												 // store for next iteration
        state.d_preconditioner[x] = pre;

        const float p =  pre * residuum;					 // apply preconditioner M^-1
        state.d_p[x] = p;

        d = residuum * p;								 // x-th term of nomimator for computing alpha and denominator for computing beta
        
    }
    
    d = warpReduce(d);
    if ((threadIdx.x & WARP_MASK) == 0) {
        atomicAdd(state.d_scanAlpha, d);
    }
}

__global__ void PCGInit_Kernel2(unsigned int N, SolverState state)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < N) {
        state.d_rDotzOld[x] = state.d_scanAlpha[0];
        state.d_delta[x] = 0.0;
    }
}

void Initialization(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N;

	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}
    cutilSafeCall(hipMemset(state.d_scanAlpha, 0, sizeof(float)));
    timer.startEvent("PCGInit_Kernel1");
	PCGInit_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
    timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);

        
        checkEverythingForNan(input, state);
	#endif
    #if DEBUG_PRINT_INFO
        float scanAlpha = 0.0f;
        cutilSafeCall(hipMemcpy(&scanAlpha, state.d_scanAlpha, sizeof(float), hipMemcpyDeviceToHost));
        printf("ScanAlpha: %f\n", scanAlpha);
    #endif



	timer.startEvent("PCGInit_Kernel2");
	PCGInit_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(N, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// PCG Iteration Parts
/////////////////////////////////////////////////////////////////////////

__global__ void PCGStep_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N;											// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x < N)
	{
		float tmp = 2.0*applyJTJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 
        
		state.d_Ap_X[x]  = tmp;														// store for next kernel call

		d = state.d_p[x] * tmp;													// x-th term of denominator of alpha
	}

    d = warpReduce(d);
    if ((threadIdx.x & WARP_MASK) == 0) {
        atomicAdd(state.d_scanAlpha, d); // sum over x-th terms to compute denominator of alpha inside this block
    }		
}

__global__ void PCGStep_Kernel2(SolverInput input, SolverState state)
{
	const unsigned int N = input.N;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    const float dotProduct = state.d_scanAlpha[0];

	float b = 0.0f;
	if (x < N)
	{
		float alpha = 0.0f;
		if (dotProduct > FLOAT_EPSILON) alpha = state.d_rDotzOld[x] / dotProduct;  // update step size alpha

		state.d_delta[x]  = state.d_delta[x]  + alpha*state.d_p[x];				// do a decent step

		float r = state.d_r[x] - alpha*state.d_Ap_X[x];					// update residuum
		state.d_r[x] = r;													// store for next kernel call

		float z = state.d_preconditioner[x] * r;								// apply preconditioner M^-1
		state.d_z[x] = z;													// save for next kernel call

        b = z * r;														// compute x-th term of the nominator of beta

	}


    b = warpReduce(b);
    if ((threadIdx.x & WARP_MASK) == 0) {
        atomicAdd(state.d_scanBeta, b); // sum over x-th terms to compute denominator of alpha inside this block
    }

}

__global__ void PCGStep_Kernel3(SolverInput input, SolverState state)
{
	const unsigned int N = input.N;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;


	if (x < N)
	{
        const float rDotzNew = state.d_scanBeta[0];										// get new nominator
		const float rDotzOld = state.d_rDotzOld[x];								// get old denominator

		float beta = 0.0f;
		if (rDotzOld > FLOAT_EPSILON) beta = rDotzNew / rDotzOld;					// update step size beta

		state.d_rDotzOld[x] = rDotzNew;												// save new rDotz for next iteration
		state.d_p[x]  = state.d_z[x]  + beta*state.d_p[x];							// update decent direction
	}
}



void PCGIteration(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N;	// Number of block variables

	// Do PCG step
	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}
    cutilSafeCall(hipMemset(state.d_scanAlpha, 0, sizeof(float)));
    timer.startEvent("PCGStep_Kernel1");
    PCGStep_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK>> >(input, state, parameters);
    timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);

        
        checkEverythingForNan(input, state);
	#endif
    #if DEBUG_PRINT_INFO
        float scanAlpha = 0.0f;
        cutilSafeCall(hipMemcpy(&scanAlpha, state.d_scanAlpha, sizeof(float), hipMemcpyDeviceToHost));
        printf("ScanAlpha: %f\n", scanAlpha);
    #endif
    
    cutilSafeCall(hipMemset(state.d_scanBeta, 0, sizeof(float)));
	timer.startEvent("PCGStep_Kernel2");
	PCGStep_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK>> >(input, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);

        
        checkEverythingForNan(input, state);
	#endif
    #if DEBUG_PRINT_INFO
        float scanBeta = 0.0f;
        cutilSafeCall(hipMemcpy(&scanBeta, state.d_scanBeta, sizeof(float), hipMemcpyDeviceToHost));
        printf("ScanBeta: %f\n", scanBeta);
    #endif


	timer.startEvent("PCGStep_Kernel3");
	PCGStep_Kernel3 << <blocksPerGrid, THREADS_PER_BLOCK>> >(input, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
        checkEverythingForNan(input, state);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// Apply Update
/////////////////////////////////////////////////////////////////////////

__global__ void ApplyLinearUpdateDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		state.d_x[x] = state.d_x[x] + state.d_delta[x];
	}
}

void ApplyLinearUpdate(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N; // Number of block variables
    timer.startEvent("ApplyLinearUpdateDevice");
	ApplyLinearUpdateDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
    timer.endEvent();
	cutilSafeCall(hipDeviceSynchronize()); // Hm

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////
// Main GN Solver Loop
////////////////////////////////////////////////////////////////////

extern "C" void solveSFSStub(SolverInput& input, SolverState& state, SolverParameters& parameters, ConvergenceAnalysis<float>* ca)
{
    CUDATimer timer;

    timer.reset();
    parameters.weightShading = parameters.weightShadingStart;

	for (unsigned int nIter = 0; nIter < parameters.nNonLinearIterations; nIter++)
	{
		float residual = EvalResidual(input, state, parameters, timer);
		printf("%i: cost: %f\n", nIter, residual);

		Initialization(input, state, parameters, timer);

		for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++) {
            printf("Iteration %d\n", linIter);
			PCGIteration(input, state, parameters, timer);
            parameters.weightShading += parameters.weightShadingIncrement;
            if (ca != NULL) 
                ca->addSample(FunctionValue<float>(EvalResidual(input, state, parameters, timer)));
		}

		ApplyLinearUpdate(input, state, parameters, timer);	//this should be also done in the last PCGIteration

        timer.nextIteration();

	}
    timer.evaluate();


	float residual = EvalResidual(input, state, parameters, timer);
	printf("final cost: %f\n", residual);
}

__global__ void PCGStep_Kernel_SaveInitialCostJTFAndPre(SolverInput input, SolverState state, SolverParameters parameters,
    float* costResult, float* jtfResult, float* preResult) {

    const unsigned int N = input.N;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < N)
    {
        float pre = 1.0f;
        costResult[x] = evalFDevice(x, input, state, parameters);
        
        const float residuum = evalMinusJTFDevice(x, input, state, parameters, pre); // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 
        jtfResult[x] = -2.0f*residuum;//TODO: port
        preResult[x] = pre;
    }

}

__global__ void PCGStep_Kernel_SaveJTJ(SolverInput input, SolverState state, SolverParameters parameters, float* jtjResult)
{
    const unsigned int N = input.N;											// Number of block variables
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < N)
    {
        jtjResult[x] = 2.0f * applyJTJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 
    }
}


void NonPatchSaveInitialCostJTFAndPreAndJTJ(SolverInput& input, SolverState& state, SolverParameters& parameters, float* costResult, float* jtfResult, float* preResult, float* jtjResult)
{
    const unsigned int N = input.N; // Number of block variables
    cutilSafeCall(hipDeviceSynchronize());
    PCGStep_Kernel_SaveInitialCostJTFAndPre<< <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters, costResult, jtfResult, preResult);

    cutilSafeCall(hipDeviceSynchronize());
    cutilCheckMsg(__FUNCTION__);

    CUDATimer timer;
    timer.reset();
    Initialization(input, state, parameters, timer);
    PCGStep_Kernel_SaveJTJ<< <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters, jtjResult);

}


extern "C" void solveSFSEvalCurrentCostJTFPreAndJTJStub(SolverInput& input, SolverState& state, SolverParameters& parameters, float* costResult, float* jtfResult, float* preResult, float* jtjResult)
{
    parameters.weightShading = parameters.weightShadingStart;


    NonPatchSaveInitialCostJTFAndPreAndJTJ(input, state, parameters, costResult, jtfResult, preResult, jtjResult);

}