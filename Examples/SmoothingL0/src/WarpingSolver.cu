#include "hip/hip_runtime.h"
#include <iostream>

#include "WarpingSolverParameters.h"
#include "WarpingSolverState.h"
#include "WarpingSolverUtil.h"
#include "WarpingSolverEquations.h"

#include <assert.h>
#include <stdio.h>
#include <stdint.h>

#include "CUDATimer.h"

#ifdef _WIN32
#include <conio.h>
#endif

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

/////////////////////////////////////////////////////////////////////////
// Eval Residual
/////////////////////////////////////////////////////////////////////////

__global__ void ResetResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x == 0) state.d_sumResidual[0] = 0.0f;
}

__global__ void EvalResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N)
	{
		float residual = evalFDevice(x, input, state, parameters);
		residual = warpReduce(residual);

		unsigned int laneid;
		//This command gets the lane ID within the current warp
		asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
		if (laneid == 0) {
			atomicAdd(&state.d_sumResidual[0], residual);
		}
	}
}

float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	float residual = 0.0f;

	const unsigned int N = input.N; // Number of block variables
	ResetResidualDevice << < 1, 1, 1 >> >(input, state, parameters);
	cutilSafeCall(hipDeviceSynchronize());
	timer.startEvent("EvalResidual");
	EvalResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
	timer.endEvent();
	cutilSafeCall(hipDeviceSynchronize());

	residual = state.getSumResidual();

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	return residual;
}

// For the naming scheme of the variables see:
// http://en.wikipedia.org/wiki/Conjugate_gradient_method
// This code is an implementation of their PCG pseudo code

__global__ void PCGInit_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x < N)
	{
		mat3x1 aux[4]; aux[0] = mat3x1(state.d_auxFloat3CM[x]); aux[1] = mat3x1(state.d_auxFloat3CP[x]); aux[2] = mat3x1(state.d_auxFloat3MC[x]); aux[3] = mat3x1(state.d_auxFloat3PC[x]);

		const float3 residuum = evalMinusJTFDevice(x, aux, input, state, parameters); // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 
		state.d_r[x]  = residuum;												 // store for next iteration

		const float3 p  = state.d_precondioner[x]  * residuum;					 // apply preconditioner M^-1
		state.d_p[x] = p;

		d = dot(residuum, p);								 // x-th term of nomimator for computing alpha and denominator for computing beta
	}
	else
	{
		state.d_p[x] = make_float3(0.0f, 0.0f, 0.0f);
	}

	bucket[threadIdx.x] = d;

	scanPart1(threadIdx.x, blockIdx.x, blockDim.x, state.d_scanAlpha);		// sum over x-th terms to compute nominator and denominator of alpha and beta inside this block
}

__global__ void PCGInit_Kernel2(unsigned int N, SolverState state)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	scanPart2(threadIdx.x, blockDim.x, gridDim.x, state.d_scanAlpha);		// sum over block results to compute nominator and denominator of alpha and beta

	if (x < N) state.d_rDotzOld[x] = bucket[0];								// store result for next kernel call
}

void Initialization(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N;

	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	const int shmem_size = sizeof(float)*THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

    timer.startEvent("PCGInit_Kernel1");
	PCGInit_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state, parameters);
    timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	timer.startEvent("PCGInit_Kernel2");
	PCGInit_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(N, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// PCG Iteration Parts
/////////////////////////////////////////////////////////////////////////

__global__ void PCGStep_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N;											// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x < N)
	{
		const float3 tmp = applyJTJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 

		state.d_Ap_X[x]  = tmp;														// store for next kernel call

		d = dot(state.d_p[x], tmp);													// x-th term of denominator of alpha
	}

	bucket[threadIdx.x] = d;

	scanPart1(threadIdx.x, blockIdx.x, blockDim.x, state.d_scanAlpha);		// sum over x-th terms to compute denominator of alpha inside this block
}

__global__ void PCGStep_Kernel2(SolverInput input, SolverState state)
{
	const unsigned int N = input.N;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	scanPart2(threadIdx.x, blockDim.x, gridDim.x, state.d_scanAlpha);		// sum over block results to compute denominator of alpha
	const float dotProduct = bucket[0];

	float b = 0.0f;
	if (x < N)
	{
		float alpha = 0.0f;
		if (dotProduct > FLOAT_EPSILON) alpha = state.d_rDotzOld[x] / dotProduct;  // update step size alpha

		state.d_delta[x]  = state.d_delta[x]  + alpha*state.d_p[x];				// do a decent step

		float3 r = state.d_r[x] - alpha*state.d_Ap_X[x];					// update residuum
		state.d_r[x] = r;													// store for next kernel call

		float3 z = state.d_precondioner[x] * r;								// apply preconditioner M^-1
		state.d_z[x] = z;													// save for next kernel call

		b = dot(z, r);														// compute x-th term of the nominator of beta
	}

	__syncthreads();														// Only write if every thread in the block has has read bucket[0]

	bucket[threadIdx.x] = b;

	scanPart1(threadIdx.x, blockIdx.x, blockDim.x, state.d_scanBeta);		// sum over x-th terms to compute nominator of beta inside this block
}

__global__ void PCGStep_Kernel3(SolverInput input, SolverState state)
{
	const unsigned int N = input.N;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	scanPart2(threadIdx.x, blockDim.x, gridDim.x, state.d_scanBeta);		// sum over block results to compute nominator of beta

	if (x < N)
	{
		const float rDotzNew = bucket[0];										// get new nominator
		const float rDotzOld = state.d_rDotzOld[x];								// get old denominator

		float beta = 0.0f;
		if (rDotzOld > FLOAT_EPSILON) beta = rDotzNew / rDotzOld;					// update step size beta

		state.d_rDotzOld[x] = rDotzNew;												// save new rDotz for next iteration
		state.d_p[x]  = state.d_z[x]  + beta*state.d_p[x];							// update decent direction
	}
}

void PCGIteration(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N;	// Number of block variables

	// Do PCG step
	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	const int shmem_size = sizeof(float)*THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

    timer.startEvent("PCGStep_Kernel1");
    PCGStep_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state, parameters);
    timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	timer.startEvent("PCGStep_Kernel2");
	PCGStep_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	timer.startEvent("PCGStep_Kernel3");
	PCGStep_Kernel3 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// Apply Update
/////////////////////////////////////////////////////////////////////////

__global__ void ApplyLinearUpdateDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		state.d_x[x] = state.d_x[x] + state.d_delta[x];
	}
}

void ApplyLinearUpdate(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N; // Number of block variables
    timer.startEvent("ApplyLinearUpdateDevice");
	ApplyLinearUpdateDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
    timer.endEvent();
	cutilSafeCall(hipDeviceSynchronize()); // Hm

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// Update Aux
/////////////////////////////////////////////////////////////////////////

__global__ void UpdateAuxDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N)
	{
		mat3x1 aux[4]; aux[0].setZero(); aux[1].setZero(); aux[2].setZero(); aux[3].setZero();
		const float2 offsets[4] = { make_float2(0, -1), make_float2(0, 1), make_float2(-1, 0), make_float2(1, 0), };
		
		mat3x1 p = mat3x1(state.d_x[x]);
		
		for (unsigned int k = 0; k < 4; k++)
		{
			int i; int j; get2DIdx(x, input.width, input.height, i, j); const int n_i = i + offsets[k].x; const int n_j = j + offsets[k].y;
			if (isInsideImage(n_i, n_j, input.width, input.height))
			{
				mat3x1 q = mat3x1(state.d_x[get1DIdx(n_i, n_j, input.width, input.height)]);
				mat3x1 d = p - q;
				float  v = d.getTranspose()*d;
		
				if (v < parameters.weightRegularizer / parameters.weightBeta) aux[k].setZero();
				else														  aux[k] = d;
			}
		}

		state.d_auxFloat3CM[x] = aux[0]; state.d_auxFloat3CP[x] = aux[1]; state.d_auxFloat3MC[x] = aux[2]; state.d_auxFloat3PC[x] = aux[3];
	}
}

void UpdateAux(SolverInput& input, SolverState& state, SolverParameters& parameters)
{
	const unsigned int N = input.N;
	UpdateAuxDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////
// Main GN Solver Loop
////////////////////////////////////////////////////////////////////

extern "C" void ImageWarpiungSolveGNStub(SolverInput& input, SolverState& state, SolverParameters& parameters)
{
    CUDATimer timer;

	UpdateAux(input, state, parameters);

	for (unsigned int nIter = 0; nIter < parameters.nNonLinearIterations; nIter++)
	{
		float residual = EvalResidual(input, state, parameters, timer);
		printf("%i: cost: %f\n", nIter, residual);

		Initialization(input, state, parameters, timer);

		for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++) {
			PCGIteration(input, state, parameters, timer);
		}

		ApplyLinearUpdate(input, state, parameters, timer);	//this should be also done in the last PCGIteration

		UpdateAux(input, state, parameters);

		if (parameters.weightBeta < 1024 * 1024) parameters.weightBeta *= 2.0f;

        timer.nextIteration();
	}
    timer.evaluate();

	float residual = EvalResidual(input, state, parameters, timer);
	printf("final cost: %f\n", residual);
}
