#include "hip/hip_runtime.h"
#include <iostream>

#include "WarpingSolverParameters.h"
#include "WarpingSolverState.h"
#include "WarpingSolverUtil.h"
#include "WarpingSolverEquations.h"

#include <assert.h>
#include <stdio.h>
#include <stdint.h>

#include "CUDATimer.h"

#ifdef _WIN32
#include <conio.h>
#endif

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

/////////////////////////////////////////////////////////////////////////
// Eval Residual
/////////////////////////////////////////////////////////////////////////

__global__ void ResetResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x == 0) state.d_sumResidual[0] = 0.0f;
}

__global__ void EvalResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N)
	{
		float4 residual = evalFDevice(x, input, state, parameters);
		float r = warpReduce(residual.x + residual.y + residual.z + residual.w);
		unsigned int laneid;
		//This command gets the lane ID within the current warp
		asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
		if (laneid == 0) {
			atomicAdd(&state.d_sumResidual[0], r);
		}
	}
}

float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	float residual = 0.0f;

	const unsigned int N = input.N; // Number of block variables
	ResetResidualDevice << < 1, 1, 1 >> >(input, state, parameters);
	cutilSafeCall(hipDeviceSynchronize());
	timer.startEvent("EvalResidual");
	EvalResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
	timer.endEvent();
	cutilSafeCall(hipDeviceSynchronize());

	residual = state.getSumResidual();

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	return residual;
}

// For the naming scheme of the variables see:
// http://en.wikipedia.org/wiki/Conjugate_gradient_method
// This code is an implementation of their PCG pseudo code

__global__ void PCGInit_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x < N)
	{
		const float4 residuum = evalMinusJTFDevice(x, input, state, parameters); // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 
		state.d_r[x]  = residuum;												 // store for next iteration

		const float4 p  = state.d_precondioner[x]  * residuum;					 // apply preconditioner M^-1
		state.d_p[x] = p;

		d = dot(residuum, p);								 // x-th term of nomimator for computing alpha and denominator for computing beta
	}
	else
	{
		state.d_p[x] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	}

	bucket[threadIdx.x] = d;

	scanPart1(threadIdx.x, blockIdx.x, blockDim.x, state.d_scanAlpha);		// sum over x-th terms to compute nominator and denominator of alpha and beta inside this block
}

__global__ void PCGInit_Kernel2(unsigned int N, SolverState state)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	scanPart2(threadIdx.x, blockDim.x, gridDim.x, state.d_scanAlpha);		// sum over block results to compute nominator and denominator of alpha and beta

	if (x < N) state.d_rDotzOld[x] = bucket[0];								// store result for next kernel call
}

void Initialization(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N;

	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	const int shmem_size = sizeof(float)*THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

    timer.startEvent("PCGInit_Kernel1");
	PCGInit_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state, parameters);
    timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	timer.startEvent("PCGInit_Kernel2");
	PCGInit_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(N, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// PCG Iteration Parts
/////////////////////////////////////////////////////////////////////////

__global__ void PCGStep_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N;											// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x < N)
	{
		const float4 tmp = applyJTJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 

		state.d_Ap_X[x]  = tmp;														// store for next kernel call

		d = dot(state.d_p[x], tmp);													// x-th term of denominator of alpha
	}

	bucket[threadIdx.x] = d;

	scanPart1(threadIdx.x, blockIdx.x, blockDim.x, state.d_scanAlpha);		// sum over x-th terms to compute denominator of alpha inside this block
}

__global__ void PCGStep_Kernel2(SolverInput input, SolverState state)
{
	const unsigned int N = input.N;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	scanPart2(threadIdx.x, blockDim.x, gridDim.x, state.d_scanAlpha);		// sum over block results to compute denominator of alpha
	const float dotProduct = bucket[0];

	float b = 0.0f;
	if (x < N)
	{
		float alpha = 0.0f;
		if (dotProduct > FLOAT_EPSILON) alpha = state.d_rDotzOld[x] / dotProduct;  // update step size alpha

		state.d_delta[x]  = state.d_delta[x]  + alpha*state.d_p[x];				// do a decent step

		float4 r = state.d_r[x] - alpha*state.d_Ap_X[x];					// update residuum
		state.d_r[x] = r;													// store for next kernel call

		float4 z = state.d_precondioner[x] * r;								// apply preconditioner M^-1
		state.d_z[x] = z;													// save for next kernel call

		b = dot(z, r);														// compute x-th term of the nominator of beta
	}

	__syncthreads();														// Only write if every thread in the block has has read bucket[0]

	bucket[threadIdx.x] = b;

	scanPart1(threadIdx.x, blockIdx.x, blockDim.x, state.d_scanBeta);		// sum over x-th terms to compute nominator of beta inside this block
}

__global__ void PCGStep_Kernel3(SolverInput input, SolverState state)
{
	const unsigned int N = input.N;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	scanPart2(threadIdx.x, blockDim.x, gridDim.x, state.d_scanBeta);		// sum over block results to compute nominator of beta

	if (x < N)
	{
		const float rDotzNew = bucket[0];										// get new nominator
		const float rDotzOld = state.d_rDotzOld[x];								// get old denominator

		float beta = 0.0f;
		if (rDotzOld > FLOAT_EPSILON) beta = rDotzNew / rDotzOld;					// update step size beta

		state.d_rDotzOld[x] = rDotzNew;												// save new rDotz for next iteration
		state.d_p[x]  = state.d_z[x]  + beta*state.d_p[x];							// update decent direction
	}
}

void PCGIteration(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N;	// Number of block variables

	// Do PCG step
	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	const int shmem_size = sizeof(float)*THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

    timer.startEvent("PCGStep_Kernel1");
    PCGStep_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state, parameters);
    timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	timer.startEvent("PCGStep_Kernel2");
	PCGStep_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif

	timer.startEvent("PCGStep_Kernel3");
	PCGStep_Kernel3 << <blocksPerGrid, THREADS_PER_BLOCK, shmem_size >> >(input, state);
	timer.endEvent();
	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

/////////////////////////////////////////////////////////////////////////
// Apply Update
/////////////////////////////////////////////////////////////////////////

__global__ void ApplyLinearUpdateDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.N; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		state.d_x[x] = state.d_x[x] + state.d_delta[x];
	}
}

void ApplyLinearUpdate(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer& timer)
{
	const unsigned int N = input.N; // Number of block variables
    timer.startEvent("ApplyLinearUpdateDevice");
	ApplyLinearUpdateDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
    timer.endEvent();
	cutilSafeCall(hipDeviceSynchronize()); // Hm

	#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
	#endif
}

////////////////////////////////////////////////////////////////////
// Main GN Solver Loop
////////////////////////////////////////////////////////////////////

extern "C" void ImageWarpiungSolveGNStub(SolverInput& input, SolverState& state, SolverParameters& parameters)
{
    CUDATimer timer;
	printf("residual=%f\n", EvalResidual(input, state, parameters, timer));

	for (unsigned int nIter = 0; nIter < parameters.nNonLinearIterations; nIter++)
	{
		Initialization(input, state, parameters, timer);

		for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++) {
			PCGIteration(input, state, parameters, timer);
		}

		ApplyLinearUpdate(input, state, parameters, timer);	//this should be also done in the last PCGIteration
		printf("residual=%f\n", EvalResidual(input, state, parameters, timer));

        timer.nextIteration();
	}
    timer.evaluate();
}
